
#include <iostream>
#include "elas.h"
#include "elas_gpu.h"
#include "image.h"

using namespace std;

#define N 8

int main (int argc, char** argv) {

  /*set up memory on the host*/
  float h_in[N];
  float h_out[N];
  for(int i=0; i<N; i++)
    h_in[i] = (float)i+10;

  /*set up memory on the device*/
  float *d_in, *d_out;
  hipMalloc((void**) &d_in, N*sizeof(float));
  hipMalloc((void**) &d_out, N*sizeof(float));

  /*transfer to device*/
  hipMemcpy(d_in, h_in, N*sizeof(float), hipMemcpyHostToDevice);

  /*launch kernel*/
  square<<<1, N>>>(d_out, d_in);
  hipDeviceSynchronize();

  /*transfer to host*/
  hipMemcpy(h_out, d_out, N*sizeof(float), hipMemcpyDeviceToHost);
    
  hipFree(d_in);
  hipFree(d_out);
    
  /*print results*/
  for(int i=0; i<N; i++)
    cout << "h_in[" << i << "]^2 = " << h_in[i] << "^2 = " << h_out[i] << endl;
  
  return EXIT_SUCCESS;
}


