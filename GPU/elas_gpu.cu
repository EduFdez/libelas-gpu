#include "hip/hip_runtime.h"
#include "elas_gpu.h"

using namespace std;


void ElasGPU::removeInconsistentSupportPoints (int16_t* D_can,int32_t D_can_width,int32_t D_can_height) {
  
  cout << "Lower removeInconsistentSupportPoints has been called~!!!" << endl;


  // for all valid support points do
  for (int32_t u_can=0; u_can<D_can_width; u_can++) {
    for (int32_t v_can=0; v_can<D_can_height; v_can++) {
      int16_t d_can = *(D_can+getAddressOffsetImage(u_can,v_can,D_can_width));
      //If the point has a calulated disparity
      if (d_can>=0) {
        // compute number of other points supporting the current point
        int32_t support = 0;
        //Checks a 5 pixel window for inconsistent disparities
        for (int32_t u_can_2=u_can-param.incon_window_size; u_can_2<=u_can+param.incon_window_size; u_can_2++) {
          for (int32_t v_can_2=v_can-param.incon_window_size; v_can_2<=v_can+param.incon_window_size; v_can_2++) {
            //Check we're inside candidate array (slightly smaller than image)
            if (u_can_2>=0 && v_can_2>=0 && u_can_2<D_can_width && v_can_2<D_can_height) {
              int16_t d_can_2 = *(D_can+getAddressOffsetImage(u_can_2,v_can_2,D_can_width));
              //Check if the disparity is different above a given threshold (5 pixels)
              //If it is considered fine, similar to other pixels around it, consider it a support
              if (d_can_2>=0 && abs(d_can-d_can_2)<=param.incon_threshold)
                support++;
            }
          }
        }
        
        // invalidate support point if number of supporting points is too low
        if (support<param.incon_min_support)
          *(D_can+getAddressOffsetImage(u_can,v_can,D_can_width)) = -1;
      }
    }
  }
}