#include "hip/hip_runtime.h"
#include "elas_gpu.h"

using namespace std;

__device__ uint32_t getAddressOffsetImage_GPU (const int32_t& u,const int32_t& v,const int32_t& width) {
  return v*width+u;
}

__device__ uint32_t getAddressOffsetGrid_GPU (const int32_t& x,const int32_t& y,const int32_t& d,const int32_t& width,const int32_t& disp_num) {
  return (y*width+x)*disp_num+d;
}

/**
 * CUDA Kernel for computing the match for a single UV coordinate
 */
__global__ void findMatch_GPU (int32_t* u_vals, int32_t* v_vals, int32_t size_vals, float plane_a, float plane_b, float plane_c,
                         int32_t* disparity_grid,int32_t *grid_dims,uint8_t* I1_desc,uint8_t* I2_desc,
                         int32_t* P, int32_t plane_radius, int32_t width ,int32_t height, bool valid, bool right_image, float* D) {
 
  // get image width and height
  const int32_t disp_num    = grid_dims[0]-1;
  const int32_t window_size = 2;

  
  //TODO: Remove hard code and use param
  bool subsampling = false;
  bool match_texture = false;
  int32_t grid_size = 20;

  // Pixel id
  uint32_t idx = blockDim.x*blockIdx.x + threadIdx.x;

  // Check that we are in range
  if(idx >= size_vals)
    return;

  // Else get our value
  uint32_t u = u_vals[idx];
  uint32_t v = v_vals[idx];

  // address of disparity we want to compute
  uint32_t d_addr;
  if (subsampling) d_addr = getAddressOffsetImage_GPU(u/2,v/2,width/2);
  else             d_addr = getAddressOffsetImage_GPU(u,v,width);
  
  // check if u is ok
  if (u<window_size || u>=width-window_size)
    return;

  // compute line start address
  int32_t  line_offset = 16*width*max(min(v,height-3),2);
  uint8_t *I1_line_addr,*I2_line_addr;
  if (!right_image) {
    I1_line_addr = I1_desc+line_offset;
    I2_line_addr = I2_desc+line_offset;
  } else {
    I1_line_addr = I2_desc+line_offset;
    I2_line_addr = I1_desc+line_offset;
  }

  // compute I1 block start address
  uint8_t* I1_block_addr = I1_line_addr+16*u;
  
  // does this patch have enough texture?
  int32_t sum = 0;
  for (int32_t i=0; i<16; i++)
    sum += abs((int32_t)(*(I1_block_addr+i))-128);
  if (sum<match_texture)
    return;

  // compute disparity, min disparity and max disparity of plane prior
  int32_t d_plane     = (int32_t)(plane_a*(float)u+plane_b*(float)v+plane_c);
  int32_t d_plane_min = max(d_plane-plane_radius,0);
  int32_t d_plane_max = min(d_plane+plane_radius,disp_num-1);

  // get grid pointer
  int32_t  grid_x    = (int32_t)floor((float)u/(float)grid_size);
  int32_t  grid_y    = (int32_t)floor((float)v/(float)grid_size);
  uint32_t grid_addr = getAddressOffsetGrid_GPU(grid_x,grid_y,0,grid_dims[1],grid_dims[0]);  
  int32_t  num_grid  = *(disparity_grid+grid_addr);
  int32_t* d_grid    = disparity_grid+grid_addr+1;
  
  // loop variables
  int32_t d_curr, u_warp, val;
  int32_t min_val = 10000;
  int32_t min_d   = -1;
  //__m128i xmm1    = _mm_load_si128((__m128i*)I1_block_addr);
  //__m128i xmm2;

  // left image
  if (!right_image) { 
    for (int32_t i=0; i<num_grid; i++) {
      d_curr = d_grid[i];
      if (d_curr<d_plane_min || d_curr>d_plane_max) { //If the current disparity is out of the planes range
        u_warp = u-d_curr;
        if (u_warp<window_size || u_warp>=width-window_size)
          continue;
        //updatePosteriorMinimum((__m128i*)(I2_line_addr+16*u_warp),d_curr,xmm1,xmm2,val,min_val,min_d);
        val = 0;
        for(int j=0; j<16; j++){
            val += abs((uint32_t*)(I1_block_addr+j)-(uint32_t*)(I2_line_addr+j+16*u_warp));
        }
        // xmm2 = _mm_load_si128((__m128i*)(I2_line_addr+16*u_warp));
        // xmm2 = _mm_sad_epu8(xmm1,xmm2);
        // val  = _mm_extract_epi16(xmm2,0)+_mm_extract_epi16(xmm2,4);
        if (val<min_val) {
            min_val = val;
            min_d   = d_curr;
        }
      }
    }
    //disparity inside the grid
    for (d_curr=d_plane_min; d_curr<=d_plane_max; d_curr++) {
      u_warp = u-d_curr;
      if (u_warp<window_size || u_warp>=width-window_size)
        continue;
      //   updatePosteriorMinimum((__m128i*)(I2_line_addr+16*u_warp),d_curr,valid?*(P+abs(d_curr-d_plane)):0,xmm1,xmm2,val,min_val,min_d);
      val = 0;
      for(int j=0; j<16; j++){
          val += abs((uint32_t*)(I1_block_addr+j)-(uint32_t*)(I2_line_addr+j+16*u_warp) + valid?*(P+abs(d_curr-d_plane)):0);
      }
      //   xmm2 = _mm_load_si128(I2_block_addr);
      //   xmm2 = _mm_sad_epu8(xmm1,xmm2);
      //   val  = _mm_extract_epi16(xmm2,0)+_mm_extract_epi16(xmm2,4)+w;
      if (val<min_val) {
        min_val = val;
        min_d   = d_curr;
      }
    }
    
  // right image
  } else {
    for (int32_t i=0; i<num_grid; i++) {
      d_curr = d_grid[i];
      if (d_curr<d_plane_min || d_curr>d_plane_max) {
        u_warp = u+d_curr;
        if (u_warp<window_size || u_warp>=width-window_size)
          continue;
        //updatePosteriorMinimum((__m128i*)(I2_line_addr+16*u_warp),d_curr,xmm1,xmm2,val,min_val,min_d);
        val = 0;
        for(int j=0; j<16; j++){
            val += abs((uint32_t*)(I1_block_addr+j)-(uint32_t*)(I2_line_addr+j+16*u_warp));
        }
        // xmm2 = _mm_load_si128((__m128i*)(I2_line_addr+16*u_warp));
        // xmm2 = _mm_sad_epu8(xmm1,xmm2);
        // val  = _mm_extract_epi16(xmm2,0)+_mm_extract_epi16(xmm2,4);
        if (val<min_val) {
            min_val = val;
            min_d   = d_curr;
        }
      }
    }
    for (d_curr=d_plane_min; d_curr<=d_plane_max; d_curr++) {
      u_warp = u+d_curr;
      if (u_warp<window_size || u_warp>=width-window_size)
        continue;
      //   updatePosteriorMinimum((__m128i*)(I2_line_addr+16*u_warp),d_curr,valid?*(P+abs(d_curr-d_plane)):0,xmm1,xmm2,val,min_val,min_d);
      val = 0;
      for(int j=0; j<16; j++){
          val += abs((uint32_t*)(I1_block_addr+j)-(uint32_t*)(I2_line_addr+j+16*u_warp) + valid?*(P+abs(d_curr-d_plane)):0);
      }
      //   xmm2 = _mm_load_si128(I2_block_addr);
      //   xmm2 = _mm_sad_epu8(xmm1,xmm2);
      //   val  = _mm_extract_epi16(xmm2,0)+_mm_extract_epi16(xmm2,4)+w;
      if (val<min_val) {
        min_val = val;
        min_d   = d_curr;
      }
    }
  }

  // set disparity value
  if (min_d>=0) *(D+d_addr) = min_d; // MAP value (min neg-Log probability)
  else          *(D+d_addr) = -1;    // invalid disparity
}

/**
 * This is the core method that computes the disparity of the image
 * It processes each triangle, so we create a kernel and have each thread
 * compute the matches in each triangle
 */
void ElasGPU::computeDisparity(std::vector<support_pt> p_support,std::vector<triangle> tri,int32_t* disparity_grid,int32_t *grid_dims,
                                uint8_t* I1_desc,uint8_t* I2_desc,bool right_image,float* D) {

  // number of disparities
  const int32_t disp_num  = grid_dims[0]-1;
  
  // descriptor window_size
  int32_t window_size = 2;
  
  // init disparity image to -10
  if (param.subsampling) {
    for (int32_t i=0; i<(width/2)*(height/2); i++)
      *(D+i) = -10;
  } else {
    for (int32_t i=0; i<width*height; i++)
      *(D+i) = -10;
  }
  
  // pre-compute prior 
  float two_sigma_squared = 2*param.sigma*param.sigma;
  int32_t* P = new int32_t[disp_num];
  for (int32_t delta_d=0; delta_d<disp_num; delta_d++)
    P[delta_d] = (int32_t)((-log(param.gamma+exp(-delta_d*delta_d/two_sigma_squared))+log(param.gamma))/param.beta);
  int32_t plane_radius = (int32_t)max((float)ceil(param.sigma*param.sradius),(float)2.0);

  // loop variables
  int32_t c1, c2, c3;
  float plane_a,plane_b,plane_c,plane_d;


  // CUDA copy over needed memory information
  // disparity_grid, I1_desc,I2_desc,P,D
  int32_t* d_disparity_grid, *d_grid_dims;
  int32_t* d_P;
  float* d_D;
  uint8_t* d_I1, *d_I2;
  //Allocate on global memory
  hipMalloc((void**) &d_disparity_grid, grid_dims[0]*grid_dims[1]*grid_dims[2]*sizeof(int32_t));
  hipMalloc((void**) &d_P, disp_num*sizeof(int32_t));
  hipMalloc((void**) &d_D, width*height*sizeof(float));
  hipMalloc((void**) &d_I1, 16*width*height*sizeof(uint8_t)); //Device descriptors
  hipMalloc((void**) &d_I2, 16*width*height*sizeof(uint8_t)); //Device descriptors
  hipMalloc((void**) &d_grid_dims, 3*sizeof(int32_t)); 

  //Now copy over data
  hipMemcpy(d_disparity_grid, disparity_grid, grid_dims[0]*grid_dims[1]*grid_dims[2]*sizeof(int32_t), hipMemcpyHostToDevice);
  hipMemcpy(d_P, P, disp_num*sizeof(int32_t), hipMemcpyHostToDevice);
  hipMemcpy(d_D, D, width*height*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_I1, I1_desc, 16*width*height*sizeof(uint8_t), hipMemcpyHostToDevice);
  hipMemcpy(d_I2, I2_desc, 16*width*height*sizeof(uint8_t), hipMemcpyHostToDevice);
  hipMemcpy(d_grid_dims, grid_dims, 3*sizeof(int32_t), hipMemcpyHostToDevice); 

  // for all triangles do
  for (uint32_t i=0; i<tri.size(); i++) {
    
    // get plane parameters
    uint32_t p_i = i*3;
    if (!right_image) {
      plane_a = tri[i].t1a;
      plane_b = tri[i].t1b;
      plane_c = tri[i].t1c;
      plane_d = tri[i].t2a;
    } else {
      plane_a = tri[i].t2a;
      plane_b = tri[i].t2b;
      plane_c = tri[i].t2c;
      plane_d = tri[i].t1a;
    }
    
    // triangle corners
    c1 = tri[i].c1;
    c2 = tri[i].c2;
    c3 = tri[i].c3;

    // sort triangle corners wrt. u (ascending)    
    float tri_u[3];
    if (!right_image) {
      tri_u[0] = p_support[c1].u;
      tri_u[1] = p_support[c2].u;
      tri_u[2] = p_support[c3].u;
    } else {
      tri_u[0] = p_support[c1].u-p_support[c1].d;
      tri_u[1] = p_support[c2].u-p_support[c2].d;
      tri_u[2] = p_support[c3].u-p_support[c3].d;
    }
    float tri_v[3] = {p_support[c1].v,p_support[c2].v,p_support[c3].v};
    
    for (uint32_t j=0; j<3; j++) {
      for (uint32_t k=0; k<j; k++) {
        if (tri_u[k]>tri_u[j]) {
          float tri_u_temp = tri_u[j]; tri_u[j] = tri_u[k]; tri_u[k] = tri_u_temp;
          float tri_v_temp = tri_v[j]; tri_v[j] = tri_v[k]; tri_v[k] = tri_v_temp;
        }
      }
    }
    
    // rename corners
    float A_u = tri_u[0]; float A_v = tri_v[0];
    float B_u = tri_u[1]; float B_v = tri_v[1];
    float C_u = tri_u[2]; float C_v = tri_v[2];
    
    // compute straight lines connecting triangle corners
    float AB_a = 0; float AC_a = 0; float BC_a = 0;
    if ((int32_t)(A_u)!=(int32_t)(B_u)) AB_a = (A_v-B_v)/(A_u-B_u);
    if ((int32_t)(A_u)!=(int32_t)(C_u)) AC_a = (A_v-C_v)/(A_u-C_u);
    if ((int32_t)(B_u)!=(int32_t)(C_u)) BC_a = (B_v-C_v)/(B_u-C_u);
    float AB_b = A_v-AB_a*A_u;
    float AC_b = A_v-AC_a*A_u;
    float BC_b = B_v-BC_a*B_u;
    
    // a plane is only valid if itself and its projection
    // into the other image is not too much slanted
    bool valid = fabs(plane_a)<0.7 && fabs(plane_d)<0.7;

    // Vector of all u,v pairs we need to calculate
    std::vector<std::pair<int32_t,int32_t>> to_calc;
        
    // first part (triangle corner A->B)
    if ((int32_t)(A_u)!=(int32_t)(B_u)) {
      // Starting at A_u loop till the B_u or the end of the image
      for (int32_t u=max((int32_t)A_u,0); u<min((int32_t)B_u,width); u++){
        // If we are sub-sampling skip every two
        if (!param.subsampling || u%2==0) {
          // Use linear lines, to get the bounds of where we need to check
          int32_t v_1 = (uint32_t)(AC_a*(float)u+AC_b);
          int32_t v_2 = (uint32_t)(AB_a*(float)u+AB_b);
          // Loop through these values of v and try to find the match
          for (int32_t v=min(v_1,v_2); v<max(v_1,v_2); v++)
            // If we are sub-sampling skip every two
            if (!param.subsampling || v%2==0) {
              to_calc.push_back(std::pair<int32_t,int32_t>(u,v));
            }
        }
      }
    }

    // second part (triangle corner B->C)
    if ((int32_t)(B_u)!=(int32_t)(C_u)) {
      for (int32_t u=max((int32_t)B_u,0); u<min((int32_t)C_u,width); u++){
        if (!param.subsampling || u%2==0) {
          int32_t v_1 = (uint32_t)(AC_a*(float)u+AC_b);
          int32_t v_2 = (uint32_t)(BC_a*(float)u+BC_b);
          for (int32_t v=min(v_1,v_2); v<max(v_1,v_2); v++)
            if (!param.subsampling || v%2==0) {
              to_calc.push_back(std::pair<int32_t,int32_t>(u,v));
            }
        }
      }
    }

    int size = to_calc.size()*sizeof(int32_t);
    // Convert vector to array
    int32_t* u_vals = (int32_t*)malloc(size);
    int32_t* v_vals = (int32_t*)malloc(size);

    // Save to arrays
    for(size_t j=0; j < to_calc.size(); j++) {
      u_vals[j] = to_calc.at(j).first;
      v_vals[j] = to_calc.at(j).second;
    }

    // Copy to device code
    int32_t* d_u_vals, *d_v_vals;
    hipMalloc((void**) &d_u_vals, size);
    hipMalloc((void**) &d_v_vals, size);
    hipMemcpy(d_u_vals, u_vals, size, hipMemcpyHostToDevice);
    hipMemcpy(d_v_vals, v_vals, size, hipMemcpyHostToDevice);

    // Calculate size of kernel
    int block_size = 32;
    int grid_size = 0;
    //Calculate gridsize (Add 1 if not evenly divided)
    if(to_calc.size()%block_size == 0){
        grid_size = ceil(to_calc.size()/block_size);
    }else{
        grid_size = ceil(to_calc.size()/block_size) + 1;
    }

    dim3 DimGrid(grid_size,1,1);
    dim3 DimBlock(block_size,1,1);

    // cout << "Cuda Elem Size: " << to_calc.size() << endl;
    // cout << "Cuda Block Size: " << block_size << endl;
    // cout << "Cuda Grid Size: " << grid_size << endl;

    // Next launch our CUDA kernel
    // TODO: Convert this to CUDA kernel
    // for(size_t j=0; j < to_calc.size(); j++) {
    //   int u = to_calc.at(j).first;
    //   int v = to_calc.at(j).second;
    //   // CPU Method
    //   findMatch(u,v,plane_a,plane_b,plane_c,disparity_grid,grid_dims,I1_desc,I2_desc,P,plane_radius,valid,right_image,D);
    // }

    //GPU Method
    findMatch_GPU<<<DimGrid, DimBlock>>>(d_u_vals,d_v_vals,to_calc.size(),plane_a,plane_b,plane_c,d_disparity_grid,d_grid_dims,
                                        d_I1,d_I2,d_P,plane_radius,width,height,valid,right_image,d_D);
    
    hipDeviceSynchronize();
    hipFree(d_u_vals);
    hipFree(d_v_vals);
    
  }

  // Copy the final disparity values back over
  
  hipMemcpy(disparity_grid, d_disparity_grid, grid_dims[0]*grid_dims[1]*grid_dims[2]*sizeof(int32_t), hipMemcpyDeviceToHost);
  hipMemcpy(D, d_D, width*height*sizeof(float), hipMemcpyDeviceToHost);

  
  // Free local memory
  delete[] P;


  // Free cuda memory
  hipFree(d_disparity_grid);
  hipFree(d_P);
  hipFree(d_D);
  hipFree(d_I1);
  hipFree(d_I2);
  hipFree(d_grid_dims);

}